#include "mmul.h"

void mmul(hipblasHandle_t handle, const float *A, const float *B, float *C, int n) {
  float alpha = 1.0;
  float beta = 1.0;
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, A, n, B, n, &beta, C, n);
  hipDeviceSynchronize();
}
