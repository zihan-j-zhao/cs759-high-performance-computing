#include <hip/hip_runtime.h>

#include <cmath>
#include <random>
#include <cstring>
#include <iostream>

#include "stencil.cuh"

// Reference: FAQ/BestPractices/random_numbers.md
void generate(float *output, std::size_t n, float lower, float upper) {
  std::random_device entropy_source;
  std::mt19937 generator(entropy_source());
  std::uniform_real_distribution<float> dist(lower, upper);
  for (std::size_t i = 0; i < n; ++i) {
      output[i] = dist(generator);
  }
}
// ===== above =====

int main(int argc, char *argv[]) {
  const unsigned int n = atol(argv[1]);
  const unsigned int R = atol(argv[2]);
  const unsigned int threads_per_block = atol(argv[3]);
  const unsigned int MASK_SIZE = 2 * R + 1;

  float *dImage, *dMask, *dOutput, *hImage, *hMask, *hOutput;
  hImage = new float[n];
  hMask = new float[MASK_SIZE];
  hOutput = new float[n];
  hipMalloc((void **)&dImage, n * sizeof(float));
  hipMalloc((void **)&dMask, MASK_SIZE * sizeof(float));
  hipMalloc((void **)&dOutput, n * sizeof(float));
  
  // Generate random numbers
  generate(hImage, n, -1.0, 1.0);
  generate(hMask, MASK_SIZE, -1.0, 1.0);
  hipMemcpy(dImage, hImage, n * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(dMask, hMask, MASK_SIZE * sizeof(float), hipMemcpyHostToDevice);

  // Reference: Assignments/general/timing.md
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  stencil(dImage, dMask, dOutput, n, R, threads_per_block);
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  float ms;
  hipEventElapsedTime(&ms, start, stop);
  // ===== above =====

  hipMemcpy(hOutput, dOutput, n * sizeof(float), hipMemcpyDeviceToHost);

  std::cout << hOutput[n - 1] << std::endl;
  std::cout << ms << std::endl;
  
  delete[] hImage;
  delete[] hMask;
  delete[] hOutput;
  hipFree(dImage);
  hipFree(dMask);
  hipFree(dOutput);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  return 0;
}
