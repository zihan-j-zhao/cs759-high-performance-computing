#include <hip/hip_runtime.h>

#include <cmath>
#include <random>
#include <cstring>
#include <iostream>

#include "matmul.cuh"

// Reference: FAQ/BestPractices/random_numbers.md
void generate(float *output, std::size_t n, float lower, float upper) {
  std::random_device entropy_source;
  std::mt19937 generator(entropy_source());
  std::uniform_real_distribution<float> dist(lower, upper);
  for (std::size_t i = 0; i < n; ++i) {
      output[i] = dist(generator);
  }
}
// ===== above =====

int main(int argc, char *argv[]) {
  const size_t n = atol(argv[1]);
  const unsigned int threads_per_block = atol(argv[2]);

  float *dA, *dB, *dC, *hA, *hB, *hC;
  hA = new float[n * n];
  hB = new float[n * n];
  hC = new float[n * n];
  hipMalloc((void **)&dA, n * n * sizeof(float));
  hipMalloc((void **)&dB, n * n * sizeof(float));
  hipMalloc((void **)&dC, n * n * sizeof(float));
  
  // Generate random numbers
  generate(hA, n * n, -1.0, 1.0);
  generate(hB, n * n, -1.0, 1.0);
  hipMemcpy(dA, hA, n * n * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(dB, hB, n * n * sizeof(float), hipMemcpyHostToDevice);

  // Reference: Assignments/general/timing.md
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  matmul(dA, dB, dC, n, threads_per_block);
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  float ms;
  hipEventElapsedTime(&ms, start, stop);
  // ===== above =====

  hipMemcpy(hC, dC, n * n * sizeof(float), hipMemcpyDeviceToHost);

  std::cout << hC[n * n - 1] << std::endl;
  std::cout << ms << std::endl;

  delete[] hA;
  delete[] hB;
  delete[] hC;
  hipFree(dA);
  hipFree(dB);
  hipFree(dC);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  return 0;
}
